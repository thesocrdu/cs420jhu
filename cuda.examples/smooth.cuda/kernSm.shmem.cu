/*******************************************************************************
*
*  kernelSmoother
*
*  This provides a CUDA implementation of a kernel smooother.
*   http://en.wikipedia.org/wiki/Kernel_smoother
*  The particular kernel in this file is a nearest neighbor smoother
*  in order to keep the code as simple to understand as possible.
*
*  This is implemeneted for 2-d square grids.
*
*  Parameters of note are all in struct CUDAGrid.
*    gridWidth -- size of the grid is gridWidth^2
*    kernelWidth -- region around point x,y to smooth
*        must be odd, i.e. 2k+1 smooths box with corners (x-k,y-k) to (x+k,y+k)
*    blockWidth -- number of processors per block.
*        must be ((cg.gridWidth-(cg.kernelWidth-1))^2 % (blockWidth^2)) == 0 
*        i.e. the smoothed regions must be of blocksize increments.
*
*  The smoothed region is only defined for the interior that has the kernel
*   defined inside the boundary, e.g. for gridWidth=10, kernelWidth=2 the
*   region from 2,2 to 7,7 will be smoothed. 
*
********************************************************************************/

/*******************************************************************************
*
*  CUDA concepts
*
*  This file shows how to use many features of CUDA:
*     2d grids
*     pitch allocation
*     shared memory
*
********************************************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>


// Need these at compile time now...yuck!
//   the 2-d array inside CUDA must be sized.
const unsigned GridWidth = 4112;
const unsigned KernelWidth = 17;
const unsigned BlockWidth = 16;

//
// CUDAGrid: structure to define geometry parameter.
//   set one of these up in main()
//
typedef struct
{
  unsigned gridWidth;
  unsigned kernelWidth;
  unsigned blockWidth;
} CUDAGrid;


/*------------------------------------------------------------------------------
* Name: NNSmoothKernel
* Action:  The CUDA kernel that implements kernel smoothing.
*             Yuck, that's two senses of kernel.
*-----------------------------------------------------------------------------*/
__global__ void NNSmoothKernel ( float* pFieldIn, float* pFieldOut, size_t pitch, CUDAGrid cg )
{ 
  extern __shared__ float shared[][BlockWidth+KernelWidth-1];

  // pitch is in bytes, figure out the number of elements for addressing
  unsigned pitchels = pitch/sizeof(float);

  // compute the halfwidth-1 of the kernel
  unsigned koffset = (cg.kernelWidth-1)/2;


  // Construct the 2d shared memory array it needs to be blockWidth+(kernelWidth-1)/2 square
  // Each node loads one element
  shared[threadIdx.x][threadIdx.y] = 
    pFieldIn [  (blockIdx.y * blockDim.y + threadIdx.y) * pitchels 
                   +  blockIdx.x * blockDim.x + threadIdx.x ];

  // And determines if it needs to load it's x-neigbor
  if ( threadIdx.x < cg.kernelWidth -1 )
  {
    shared[threadIdx.x + cg.blockWidth][threadIdx.y] = 
      pFieldIn [  (blockIdx.y * blockDim.y + threadIdx.y) * pitchels 
                     +  blockIdx.x * blockDim.x + threadIdx.x + cg.blockWidth ];
  }

  // And determines if it needs to load it's y-neigbor
  if ( threadIdx.y < cg.kernelWidth -1 )
  {
    shared[threadIdx.x][threadIdx.y + cg.blockWidth] = 
      pFieldIn [  (blockIdx.y * blockDim.y + threadIdx.y + cg.blockWidth) * pitchels 
                     +  blockIdx.x * blockDim.x + threadIdx.x];
  }

  // And determines if it needs to load it's xy-neigbor
  if ( ( threadIdx.y < cg.kernelWidth -1 ) && ( threadIdx.x < cg.kernelWidth -1 ))
  {
    shared[threadIdx.x + cg.blockWidth][threadIdx.y + cg.blockWidth] = 
      pFieldIn [  (blockIdx.y * blockDim.y + threadIdx.y + cg.blockWidth) * pitchels 
                     +  blockIdx.x * blockDim.x + threadIdx.x + cg.blockWidth];
  }

  __syncthreads();

  pFieldOut [ (threadIdx.y+koffset)*pitchels + threadIdx.x+koffset ] = shared [threadIdx.x][threadIdx.y];


  // Variable to accumulate the smoothed value
  float value = 0.0;

  // The grid indexes start from 
  unsigned xindex = ( blockIdx.x * blockDim.x + threadIdx.x) + koffset; 
  unsigned yindex = ( blockIdx.y * blockDim.y + threadIdx.y) + koffset; 

  // Get the value from the kernel
  for ( unsigned j=0; j<cg.kernelWidth; j++ )
  {
    for ( unsigned i=0; i<cg.kernelWidth; i++ )
    {
      value += shared [threadIdx.x+i] [threadIdx.y+j];
    }
  }
  
  // Divide by the number of elements in the kernel
  value /= cg.kernelWidth*cg.kernelWidth;

  // Write the value out 
  pFieldOut [ yindex*pitchels + xindex ] = value;


} 


/*------------------------------------------------------------------------------
* Name:  SmoothField
* Action:  Host entry point to kernel smoother
*-----------------------------------------------------------------------------*/
bool SmoothField ( float* pHostFieldIn, float *pHostFieldOut, CUDAGrid cg ) 
{ 
  float * pDeviceFieldIn = 0;
  float * pDeviceFieldOut = 0;

  size_t pitch, pitchout;

  struct timeval ta, tb, tc, td;

  // Check the grid dimensions and extract parameters.  See top description about restrictions
  assert ((( cg.kernelWidth -1 )%2) == 0 );     // Width is odd
  unsigned blockSize = cg.blockWidth * cg.blockWidth;  
  assert( ((cg.gridWidth-(cg.kernelWidth-1))*(cg.gridWidth-(cg.kernelWidth-1)) % blockSize) == 0 );

  gettimeofday ( &ta, NULL );

  // Place the data set on device memory
  hipMallocPitch((void**)&pDeviceFieldIn, &pitch, cg.gridWidth*sizeof(float), cg.gridWidth ); 
  hipMemcpy2D ( pDeviceFieldIn, pitch,
                 pHostFieldIn, cg.gridWidth*sizeof(float), cg.gridWidth*sizeof(float), cg.gridWidth,
                 hipMemcpyHostToDevice); 

  // Allocate the output
  hipMallocPitch((void**)&pDeviceFieldOut, &pitchout, cg.gridWidth*sizeof(float), cg.gridWidth ); 

  gettimeofday ( &tb, NULL );

  // Construct a 2d grid/block
  const dim3 DimBlock ( cg.blockWidth, cg.blockWidth );
  const dim3 DimGrid ( (cg.gridWidth-(cg.kernelWidth-1))/cg.blockWidth , 
                       (cg.gridWidth-(cg.kernelWidth-1))/cg.blockWidth );
  const unsigned shmemSize = ( cg.blockWidth + cg.kernelWidth -1 ) * ( cg.blockWidth + cg.kernelWidth -1 ) * sizeof (float);

  // Invoke the kernel
  NNSmoothKernel <<<DimGrid,DimBlock, shmemSize>>> ( pDeviceFieldIn, pDeviceFieldOut, pitch, cg ); 

  gettimeofday ( &tc, NULL );

  // Retrieve the results
  hipMemcpy2D(pHostFieldOut, cg.gridWidth*sizeof(float), 
               pDeviceFieldOut, pitch, cg.gridWidth*sizeof(float), cg.gridWidth,
               hipMemcpyDeviceToHost); 

  gettimeofday ( &td, NULL );


  if ( ta.tv_usec < td.tv_usec )
  {
    printf ("Elapsed total time (s/m): %d:%d\n", td.tv_sec - ta.tv_sec, td.tv_usec - ta.tv_usec );
  } else {
    printf ("Elapsed total time (s/m): %d:%d\n", td.tv_sec - ta.tv_sec - 1, 1000000 - td.tv_usec + ta.tv_usec );
  }

  if ( tb.tv_usec < tc.tv_usec )
  {
    printf ("Elapsed kernel time (s/m): %d:%d\n", tc.tv_sec - tb.tv_sec, tc.tv_usec - tb.tv_usec );
  } else {
    printf ("Elapsed kernel time (s/m): %d:%d\n", tc.tv_sec - tb.tv_sec - 1, 1000000 - tc.tv_usec + tb.tv_usec );
  }

  return true;
}



/*------------------------------------------------------------------------------
* Name:  initField
* Action:  Initialize a field to predictable values.
*    This is a useful format for debugging, because values 
*    accumulate to their initial value.
*-----------------------------------------------------------------------------*/
void initField ( unsigned dim, float* pField )
{
  for ( unsigned j=0; j<dim; j++ )
  {
    for ( unsigned i=0; i<dim; i++ )
    {
      pField[j*dim+i] = j + i;
    }
  }
}


/*------------------------------------------------------------------------------
* Name:  main
* Action:  Entry point
*-----------------------------------------------------------------------------*/
int main ()
{

  // Define the grid
  CUDAGrid cg;
  cg.gridWidth = GridWidth;
  cg.kernelWidth = KernelWidth;
  cg.blockWidth = BlockWidth;

  // Create the input field
  float *field = (float *) malloc ( cg.gridWidth * cg.gridWidth * sizeof(float));
  initField ( cg.gridWidth, field );

  // Create the output field
  float *out = (float *) malloc ( cg.gridWidth * cg.gridWidth * sizeof(float));

  // Call the kernel
  SmoothField ( field, out, cg );

  // Print the output field (for debugging purposes.
/*  unsigned koffset = (cg.kernelWidth-1)/2;
  for ( unsigned j=0; j< cg.gridWidth; j++ )
  {
    for ( unsigned i=0; i< cg.gridWidth; i++ )
    {
      if ( ( i >= koffset ) && 
           ( j >= koffset ) &&
           ( i < ( cg.gridWidth - koffset )) &&
           ( j < ( cg.gridWidth - koffset )) )
      {
        printf ("%4.0f, ", out[j*cg.gridWidth + i]);
      }
      else
      {
        printf ("  na, ");
      }
    }  
    printf ("\n");
  }
*/
}
/*******************************************************************************
*
*  Revsion History
*
*  $Log: kernSm.shmem.cu,v $
*  Revision 1.1  2009/07/22 16:54:20  randal
*  OK, got them good.
*
*  Revision 1.2  2009/07/22 15:21:41  randal
*  This is the preferred impelmentation.
*
*
*******************************************************************************/
